#include "hip/hip_runtime.h"


#include <common/cuda_tools.hpp>

namespace smc {

    namespace scrfd {

        static const int NUM_BOX_ELEMENT = 16;

        static __device__ void affine_project(float *matrix, float x, float y, float *ox, float *oy) {
            *ox = matrix[0] * x + matrix[1] * y + matrix[2];
            *oy = matrix[3] * x + matrix[4] * y + matrix[5];
        }

        static __device__ float sigmoid(float x) {
            return 1.0f / (1.0f + exp(-x));
        }

        static __global__ void decode_kernel(
                float *predict, int num_bboxes, float deconfidence_threshold, float nms_threshold,
                float *invert_affine_matrix, float *parray, int max_objects, float *prior_array
        ) {
            int position = blockDim.x * blockIdx.x + threadIdx.x;
            if (position >= num_bboxes) return;

            float *pitem = predict + 15 * position;

            // cx, cy, w, h, conf, keepflag, landmark0.x, landmark0.y, landmark1.x, landmark1.y, landmark2.x, landmark2.y
            float deconfidence = pitem[4];
            if (deconfidence < deconfidence_threshold)
                return;

            int index = atomicAdd(parray, 1);
            if (index >= max_objects)
                return;

            float *prior = prior_array + 4 * position;
            float dx = pitem[0] * prior[2];
            float dy = pitem[1] * prior[2];
            float dr = pitem[2] * prior[2];
            float db = pitem[3] * prior[2];
            float left = prior[0] - dx;
            float top = prior[1] - dy;
            float right = prior[0] + dr;
            float bottom = prior[1] + db;
            affine_project(invert_affine_matrix, left, top, &left, &top);
            affine_project(invert_affine_matrix, right, bottom, &right, &bottom);

            float *pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
            *pout_item++ = left;
            *pout_item++ = top;
            *pout_item++ = right;
            *pout_item++ = bottom;
            *pout_item++ = sigmoid(deconfidence);
            *pout_item++ = 1;  // keepflag(1=keep, 0=ignore)

            float *landmark_predict = pitem + 5;
            for (int i = 0; i < 5; ++i) {
                float x = prior[0] + landmark_predict[0] * prior[2];
                float y = prior[1] + landmark_predict[1] * prior[3];
                affine_project(invert_affine_matrix, x, y, pout_item + 0, pout_item + 1);
                pout_item += 2;
                landmark_predict += 2;
            }
        }

        static __device__ float box_iou(
                float aleft, float atop, float aright, float abottom,
                float bleft, float btop, float bright, float bbottom
        ) {

            float cleft = max(aleft, bleft);
            float ctop = max(atop, btop);
            float cright = min(aright, bright);
            float cbottom = min(abottom, bbottom);

            float c_area = max(cright - cleft, 0.0f) * max(cbottom - ctop, 0.0f);
            if (c_area == 0.0f)
                return 0.0f;

            float a_area = max(0.0f, aright - aleft) * max(0.0f, abottom - atop);
            float b_area = max(0.0f, bright - bleft) * max(0.0f, bbottom - btop);
            return c_area / (a_area + b_area - c_area);
        }

        static __global__ void nms_kernel(float *bboxes, int max_objects, float threshold) {

            int position = (blockDim.x * blockIdx.x + threadIdx.x);
            int count = min((int) *bboxes, max_objects);
            if (position >= count)
                return;

            // left, top, right, bottom, confidence, keepflag
            float *pcurrent = bboxes + 1 + position * NUM_BOX_ELEMENT;
            for (int i = 0; i < count; ++i) {
                float *pitem = bboxes + 1 + i * NUM_BOX_ELEMENT;
                if (i == position) continue;

                if (pitem[4] >= pcurrent[4]) {
                    if (pitem[4] == pcurrent[4] && i < position)
                        continue;

                    float iou = box_iou(
                            pcurrent[0], pcurrent[1], pcurrent[2], pcurrent[3],
                            pitem[0], pitem[1], pitem[2], pitem[3]
                    );

                    if (iou > threshold) {
                        pcurrent[5] = 0;  // 1=keep, 0=ignore
                        return;
                    }
                }
            }
        }

        static float desigmoid(float x) {
            return -log(1.0f / x - 1.0f);
        }

        void decode_kernel_invoker(
                float *predict, int num_bboxes, float confidence_threshold, float nms_threshold,
                float *invert_affine_matrix, float *parray, int max_objects, float *prior,
                hipStream_t stream
        ) {
            auto grid = CUDATools::grid_dims(num_bboxes);
            auto block = CUDATools::block_dims(num_bboxes);
            checkCudaKernel(decode_kernel<<<grid, block, 0, stream>>>(
                    predict, num_bboxes, desigmoid(confidence_threshold), nms_threshold,
                    invert_affine_matrix, parray, max_objects, prior
            ));

            grid = CUDATools::grid_dims(max_objects);
            block = CUDATools::block_dims(max_objects);
            checkCudaKernel(nms_kernel<<<grid, block, 0, stream>>>(parray, max_objects, nms_threshold));
        }
    };
}